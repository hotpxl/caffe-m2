#include "hip/hip_runtime.h"
// Copyright 2013 Yangqing Jia

#include <algorithm>
#include <cmath>
#include <cstdlib>
#include <cstring>

#include "caffe/common.hpp"
#include "caffe/util/im2col.hpp"

namespace caffe {

template <typename Dtype>
__global__ void im2col_gpu_kernel(const int n, const Dtype* data_im, const int num,
    const int height, const int width, const int ksize, const int channelNum, const int pad,
    const int stride, const int height_col, const int width_col,
    Dtype* data_col) {
  // CUDA_KERNEL_LOOP(index, n) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (n <= index) {
    return;
  }
  int w_out = index % width_col;
  int h_out = index / width_col % height_col;
  int imageIdx = index / width_col / height_col % num;
  int channel_in = index / width_col / height_col / num;
  int channel_out = channel_in * ksize * ksize;
  int h_in = h_out * stride - pad;
  int w_in = w_out * stride - pad;
  data_col += ((channel_out * num + imageIdx) * height_col + h_out) * width_col + w_out;
  data_im += ((imageIdx * channelNum + channel_in) * height + h_in) * width + w_in;
  for (int i = 0; i < ksize; ++i) {
    for (int j = 0; j < ksize; ++j) {
      int h = h_in + i;
      int w = w_in + j;
      *data_col = (h >= 0 && w >= 0 && h < height && w < width) ?
          data_im[i * width + j] : 0;
      data_col += height_col * width_col * num;
    }
  }
  // }
}

template <typename Dtype>
void im2col_gpu(const Dtype* data_im, const int num, const int channels,
    const int height, const int width, const int ksize, const int pad,
    const int stride, Dtype* data_col) {
  // We are going to launch channels * height_col * width_col kernels, each
  // kernel responsible for copying a single-channel grid.
  int height_col = (height + 2 * pad - ksize) / stride + 1;
  int width_col = (width + 2 * pad - ksize) / stride + 1;
  int num_kernels = num * channels * height_col * width_col;
  // NOLINT_NEXT_LINE(whitespace/operators)
  im2col_gpu_kernel<Dtype><<<CAFFE_GET_BLOCKS(num_kernels),
                             CAFFE_CUDA_NUM_THREADS>>>(
      num_kernels, data_im, num, height, width, ksize, channels, pad, stride, height_col,
      width_col, data_col);
  CUDA_POST_KERNEL_CHECK;
}


// Explicit instantiation
template void im2col_gpu<float>(const float* data_im, const int num, const int channels,
    const int height, const int width, const int ksize, const int pad,
    const int stride, float* data_col);
template void im2col_gpu<double>(const double* data_im, const int num, const int channels,
    const int height, const int width, const int ksize, const int pad,
    const int stride, double* data_col);

#if 1
template <typename Dtype>
__global__ void col2im_gpu_kernel(const int n, const Dtype* data_col, const int num,
    const int height, const int width, const int ksize, const int channelNum,
    const int pad, const int stride, const int height_col, const int width_col,
    Dtype* data_im) {
  CUDA_KERNEL_LOOP(index, n) {
    Dtype val = 0;
    int w = index % width + pad;
    int h = (index / width) % height + pad;
    int c = index / (width * height) % channelNum;
    int imageIdx = index / channelNum / width / height;
    // compute the start and end of the output
    int w_col_start = (w < ksize) ? 0 : (w - ksize) / stride + 1;
    int w_col_end = min(w / stride + 1, width_col);
    int h_col_start = (h < ksize) ? 0 : (h - ksize) / stride + 1;
    int h_col_end = min(h / stride + 1, height_col);
    /*
    for (int h_col = h_col_start; h_col < h_col_end; ++h_col) {
      for (int w_col = w_col_start; w_col < w_col_end; ++w_col) {
        // the col location: [c * width * height + h_out, w_out]
        int c_col = c * ksize * ksize + (h - h_col * stride) * ksize + (w - w_col * stride);
        val += data_col[(c_col * height_col + h_col) * width_col + w_col];
      }
    }
    */
    // equivalent implementation
    int offset = ((c * ksize * ksize + h * ksize + w) * num + imageIdx) * height_col * width_col;
    int coeff_h_col = width_col - stride * ksize * height_col * width_col * num;
    int coeff_w_col = 1 - stride * height_col * width_col * num;
    for (int h_col = h_col_start; h_col < h_col_end; ++h_col) {
      for (int w_col = w_col_start; w_col < w_col_end; ++w_col) {
        val += data_col[offset + h_col * coeff_h_col + w_col * coeff_w_col];
      }
    }
    data_im[index] = val;
  }
}
#else
template <typename T> __global__ void col2im_gpu_kernel(const int n, const T* dataCol, const int num, const int height, const int width, const int channelNum, const int ksize, const int pad, const int stride, const int heightCol, const int widthCol, T* dataImage) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int widthOffsetInCol = index % widthCol;
  int heightOffsetInCol = index / widthCol % heightCol;
  int imageIdx = index / widthCol / heightCol;
  for (int i = 0; i < channelNum * ksize * ksize; ++i) {
    int w = i % ksize;
    int h = i / ksize % ksize;
    int c = i / ksize / ksize;
    int widthOffsetInImage = widthOffsetInCol * stride + w - pad;
    int heightOffsetInImage = heightOffsetInCol * stride + h - pad;
    if (0 <= widthOffsetInImage && widthOffsetInImage < width && 0 <= heightOffsetInImage && heightOffsetInImage < height) {
      *(dataImage + (((imageIdx * channelNum + c) * height + heightOffsetInImage) * width + widthOffsetInImage)) += dataCol[(((i * num + imageIdx) * heightCol + heightOffsetInCol) * widthCol + widthOffsetInCol)];
    }
  }
}

template <> __global__ void col2im_gpu_kernel<float>(const int n, const float* dataCol, const int num, const int height, const int width, const int channelNum, const int ksize, const int pad, const int stride, const int heightCol, const int widthCol, float* dataImage) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (n <= index) {
    return;
  }
  int widthOffsetInCol = index % widthCol;
  int heightOffsetInCol = index / widthCol % heightCol;
  int imageIdx = index / widthCol / heightCol;
  int offset = (imageIdx * channelNum * height + heightOffsetInCol * stride - pad) * width + widthOffsetInCol * stride - pad;
  dataCol += (imageIdx * heightCol + heightOffsetInCol) * widthCol + widthOffsetInCol;
  for (int i = 0; i < channelNum * ksize * ksize; ++i) {
    int w = i % ksize;
    int h = i / ksize % ksize;
    int c = i / ksize / ksize;
    int widthOffsetInImage = widthOffsetInCol * stride + w - pad;
    int heightOffsetInImage = heightOffsetInCol * stride + h - pad;
    if (0 <= widthOffsetInImage && widthOffsetInImage < width && 0 <= heightOffsetInImage && heightOffsetInImage < height) {
      atomicAdd(dataImage + offset + (c * height + h) * width + w, dataCol[i * num * heightCol * widthCol]);
      // atomicAdd(dataImage + (((imageIdx * channelNum + c) * height + heightOffsetInImage) * width + widthOffsetInImage), dataCol[(((i * num + imageIdx) * heightCol + heightOffsetInCol) * widthCol + widthOffsetInCol)]);
    }
  }
}
#endif

template <typename Dtype>
void col2im_gpu(const Dtype* data_col, const int num, const int channels,
    const int height, const int width, const int ksize, const int pad,
    const int stride, Dtype* data_im) {
  // CUDA_CHECK(hipMemset(data_im, 0,
  //            sizeof(Dtype) * height * width * channels));
  int height_col = (height + 2 * pad - ksize) / stride + 1;
  int width_col = (width + 2 * pad - ksize) / stride + 1;
  int num_kernels = num * height_col * width_col;
  // To avoid involving atomic operations, we will launch one kernel per
  // bottom dimension, and then in the kernel add up the top dimensions.
  // NOLINT_NEXT_LINE(whitespace/operators)
#if 1
  col2im_gpu_kernel<Dtype><<<CAFFE_GET_BLOCKS(num_kernels),
                             CAFFE_CUDA_NUM_THREADS>>>(
      num_kernels, data_col, num, height, width, ksize, channels, pad, stride,
      height_col, width_col, data_im);
#else
  col2im_gpu_kernel<Dtype><<<CAFFE_GET_BLOCKS(num_kernels), CAFFE_CUDA_NUM_THREADS>>>(num_kernels, data_col, num, height, width, channels, ksize, pad, stride, height_col, width_col, data_im);
#endif
  CUDA_POST_KERNEL_CHECK;
}

// Explicit instantiation
template void col2im_gpu<float>(const float* data_col, const int num, const int channels,
    const int height, const int width, const int psize, const int pad,
    const int stride, float* data_im);
template void col2im_gpu<double>(const double* data_col, const int num, const int channels,
    const int height, const int width, const int psize, const int pad,
    const int stride, double* data_im);
}

